#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hipcub/hipcub.hpp>
#include <chrono>
#include <hip/hip_cooperative_groups.h>

#include "particlegrid.cuh"
#include "../Shared/cudaErrorCheck.h"

namespace cg = cooperative_groups;


constexpr int numElements = int(1e5);

template<typename itT>
void genRandomData(itT begin, itT end, int maxSize) {
    std::random_device seed;
    std::default_random_engine rng(seed());
    std::uniform_real_distribution<float> dist(0, maxSize);
    for (auto it = begin; it != end; it++) {
        *it = make_float3(dist(rng),dist(rng),dist(rng));
    }
}

inline __host__ __device__ int3 operator+(int3 a, int3 b)
{
    return make_int3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__constant__ float3 worldOrigin;
__constant__ float3 cellSize;
__constant__ uint3  gridSize;

// calculate position in uniform grid
__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floorf((p.x - worldOrigin.x) / cellSize.x);
    gridPos.y = floorf((p.y - worldOrigin.y) / cellSize.y);
    gridPos.z = floorf((p.z - worldOrigin.z) / cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (gridSize.y-1);
    gridPos.z = gridPos.z & (gridSize.z-1);
    return gridPos.z * gridSize.y * gridSize.x + gridPos.y * gridSize.x + gridPos.x;
}

// calculate grid hash value for each particle
void __global__ calcHashD(uint  *gridParticleHash,  uint   *gridParticleIndex, Particle *particles, uint numParticles)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= numParticles){
        return;
    }
    volatile float3 p = particles[index].position;

    // get address in grid
    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__
void reorderDataAndFindCellStartD(uint   *cellStart,        // output: cell start index
                                  uint   *cellEnd,          // output: cell end index
                                  Particle *sortedParticles,        // output: sorted particles
                                  uint   *gridParticleHash, // input: sorted grid hashes
                                  uint   *gridParticleIndex,// input: sorted particle indices
                                  Particle *oldParticles,       // input: particle array
                                  uint    numParticles)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = blockIdx.x *blockDim.x + threadIdx.x;

    uint hash;

    // handle case when no. of particles not multiple of block size
    if (index < numParticles)
    {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[index-1];
        }
    }

    cg::sync(cta);

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStart[hash] = index;

            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1)
        {
            cellEnd[hash] = index + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[index];
        Particle pos = oldParticles[sortedIndex];
        // float4 vel = oldVel[sortedIndex];

        sortedParticles[index] = pos;
        // sortedVel[index] = vel;
    }


}

__global__
void calcSphD(Particle *particleArray,
            Particle *oldParticles, // sorted particle array
            uint* cellStart,
            uint* cellEnd,
            uint* gridParticleIndex,
            uint numParticles)
    {
        uint index = blockIdx.x *blockDim.x + threadIdx.x;

        if (index >= numParticles) return;

        Particle part = oldParticles[index]; // Particle for which sph equations need to be computed

        // get address in grid
        int3 gridPos = calcGridPos(part.position);
    	
        //TODO: initialize values to be computed
        float density = 0.f;

        // go over all surrounding cells
        for (int z=-1; z<=1; z++)
        {
            for (int y=-1; y<=1; y++)
            {
                for (int x=-1; x<=1; x++)
                {
                    int3 neighbourPos = gridPos + make_int3(x, y, z);
                    uint gridHash = calcGridHash(neighbourPos);

                    uint startIndex = cellStart[gridHash];

                    if(startIndex != 0xffffffff) // cell not empty
                    {
                        uint endIndex = cellEnd[gridHash];

                        // TODO: Calculate SPH equations here
                        for(uint j = startIndex; j < endIndex; j++)
                        {
                            if(j != index) // exclude the particle itself from neighbors
                            {
                                Particle neighborParticle = oldParticles[j];

                                // density += neighborParticle.mass * ...

                            }
                        }
                    }
                }
            }
        }

        uint originalIndex = gridParticleIndex[index];
        // TODO: Update particle properties in the original array
        particleArray[originalIndex].density = density;
    }   

int3 calcGridPos(float3 p, float3 worldOrigin, float3 cellSize)
{
    int3 gridPos;
    gridPos.x = floorf((p.x - worldOrigin.x) / cellSize.x);
    gridPos.y = floorf((p.y - worldOrigin.y) / cellSize.y);
    gridPos.z = floorf((p.z - worldOrigin.z) / cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
uint calcGridHash(int3 gridPos, uint3 gridSize)
{
    gridPos.x = gridPos.x & (gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (gridSize.y-1);
    gridPos.z = gridPos.z & (gridSize.z-1);
    return gridPos.z * gridSize.y * gridSize.x + gridPos.y * gridSize.x + gridPos.x;
}

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


void printParticles(float3 * particles, int n){
    for (int i = 0; i < n; i++)
        std::cout << "Particle "<<particles[i].x<<", "<<particles[i].y<<", "<<particles[i].z<<std::endl;
}


void getNeighbors(float3 pos, uint* dSortedHash, uint* dSortedIndex, std::vector<uint> &neighborIndex, uint numParticles){
    // uint numThreads, numBlocks;
    // computeGridSize(numParticles, 64, numBlocks, numThreads);
    
    float3 cellSize = make_float3(2.f,2.f,2.f);
    uint3  gridSize = make_uint3(64,64,64);

    int3 gridPos;
    gridPos.x = floorf((pos.x) / cellSize.x);
    gridPos.y = floorf((pos.y) / cellSize.y);
    gridPos.z = floorf((pos.z) / cellSize.z);

    printf("(%d,%d,%d)\n", gridPos.x, gridPos.y, gridPos.z);
    for (int z=-1; z<=1; z++)
    {
        for (int y=-1; y<=1; y++)
        {
            for (int x=-1; x<=1; x++)
            {
                int3 neighborPos = gridPos + make_int3(x,y,z);
                
                neighborPos.x = neighborPos.x & (gridSize.x-1);  // wrap grid, assumes size is power of 2
                neighborPos.y = neighborPos.y & (gridSize.y-1);
                neighborPos.z = neighborPos.z & (gridSize.z-1);
                uint neighborHash = neighborPos.z * gridSize.y * gridSize.x + neighborPos.y * gridSize.x + neighborPos.x;
                // printf("\nNeighbor hash %d\n", neighborHash);
                for(uint i = 0; i<numParticles;i++){
                    if(dSortedHash[i] > neighborHash)
                        break;
                    else if(dSortedHash[i] == neighborHash){
                        // printf(" %d ", dSortedIndex[i]);
                        neighborIndex.push_back(dSortedIndex[i]);
                    }
                }
            }
        }
    }
}

void getSortedNeighbors(float3 pos, uint* cellStart, uint* cellEnd, std::vector<uint> &neighborIndex, uint numParticles){
    // uint numThreads, numBlocks;
    // computeGridSize(numParticles, 64, numBlocks, numThreads);
    
    float3 cellSize = make_float3(2.f,2.f,2.f);
    uint3  gridSize = make_uint3(64,64,64);

    int3 gridPos;
    gridPos.x = floorf((pos.x) / cellSize.x);
    gridPos.y = floorf((pos.y) / cellSize.y);
    gridPos.z = floorf((pos.z) / cellSize.z);

    printf("(%d,%d,%d)\n", gridPos.x, gridPos.y, gridPos.z);
    for (int z=-1; z<=1; z++)
    {
        for (int y=-1; y<=1; y++)
        {
            for (int x=-1; x<=1; x++)
            {
                int3 neighborPos = gridPos + make_int3(x,y,z);
                
                neighborPos.x = neighborPos.x & (gridSize.x-1);  // wrap grid, assumes size is power of 2
                neighborPos.y = neighborPos.y & (gridSize.y-1);
                neighborPos.z = neighborPos.z & (gridSize.z-1);
                uint neighborHash = neighborPos.z * gridSize.y * gridSize.x + neighborPos.y * gridSize.x + neighborPos.x;
                // printf("\nNeighbor hash %d\n", neighborHash);
                uint startIndex = cellStart[neighborHash];
                if(startIndex != 0xffffffff){
                    for(uint i = cellStart[neighborHash]; i < cellEnd[neighborHash];i++){
                        neighborIndex.push_back(i);
                        // printf(" %d ", i);
                    }
                }
             
            }
        }
    }
}



ParticleSystem::ParticleSystem(uint numParticles, float3 worldOrigin, uint3 gridSize, float h):
    m_numParticles(numParticles),
    m_particleArray(0),
    m_worldOrigin(worldOrigin),
    m_sortedParticleArray(0),
    m_gridSize(gridSize),
    m_cellSize(make_float3(2*h,2*h,2*h))
{
    m_numGridCells = gridSize.x * gridSize.y * gridSize.z;
    _init(numParticles);
}

ParticleSystem::~ParticleSystem(){
    _free();
}

void ParticleSystem::calcHash(uint  *gridParticleHash,
                uint  *gridParticleIndex,
                Particle *particles,
                int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);
    std::cout<<"Num Blocks: "<<numBlocks<<", Num Threads: "<<numThreads<<std::endl;
    // printParticles((float3 *)pos, 30);
    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,gridParticleIndex, particles, numParticles);

    gpuErrchk( hipPeekAtLastError());
}

void ParticleSystem::sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint* dSortedHash, uint* dSortedIndex, uint numParticles)
{
    // Determine temporary device storage requirements
    // this is done by passing 0 as the temp storage
    void     *tempStorage_d = NULL;
    size_t   tempStorageSize = 0;
    gpuErrchk( hipcub::DeviceRadixSort::SortPairs(tempStorage_d, tempStorageSize, dGridParticleHash,dSortedHash, dGridParticleIndex,dSortedIndex, numParticles));

    // Allocate temporary storage
    gpuErrchk( hipMalloc(&tempStorage_d, tempStorageSize));
    // Run sorting operation
    gpuErrchk( hipcub::DeviceRadixSort::SortPairs(tempStorage_d, tempStorageSize, dGridParticleHash, dSortedHash, dGridParticleIndex, dSortedIndex, numParticles));
}


void ParticleSystem::reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     Particle *sortedParticles,
                                    //  float *sortedPos,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                    //  float *oldPos,
                                     Particle *oldParticles,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        gpuErrchk(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            sortedParticles,
            gridParticleHash,
            gridParticleIndex,
            oldParticles,
            numParticles);

    }

void ParticleSystem::calcSph(Particle *particleArray, //write new properties to this array
             Particle *sortedParticles,
             uint* cellStart,
             uint* cellEnd,
             uint* gridParticleIndex,
             uint numParticles
             )         
    {

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

                // execute the kernel
        calcSphD<<< numBlocks, numThreads >>>(particleArray,
                                              sortedParticles,
                                              cellStart,
                                              cellEnd,
                                              gridParticleIndex,
                                              numParticles);

    }

void ParticleSystem::update(){

    //TODO: 
    //timeIntegration()

    calcHash(m_dGridParticleHash, m_dGridParticleIndex, m_particleArray, m_numParticles);

    gpuErrchk( hipDeviceSynchronize());
    std::cout<<"------------\n";
    for (int i = 0; i < 100; i++){
        float3 p = m_particleArray[i].position;
        std::cout << "Particle "<<p.x<<", "<<p.y<<", "<<p.z<<", "
        <<"Hash: "<<m_dGridParticleHash[i]<<
        ", Index: "<<m_dGridParticleIndex[i]<<std::endl;
    }


    sortParticles(m_dGridParticleHash, m_dGridParticleIndex, m_dSortedParticleHash, m_dSortedParticleIndex, m_numParticles);

    gpuErrchk( hipDeviceSynchronize());
    std::cout<<"------ HASHING ------\n";
    for (int i = 0; i < 100; i++){
        float3 p = m_particleArray[m_dSortedParticleIndex[i]].position;
        std::cout << "Particle "<<p.x<<", "<<p.y<<", "<<p.z
                    <<", "<<"Hash: "<<m_dSortedParticleHash[i]<<", Index: "<<m_dSortedParticleIndex[i]<<std::endl;
    }


    reorderDataAndFindCellStart(m_cellStart,
                                m_cellEnd,
                                m_sortedParticleArray,
                                m_dSortedParticleHash,
                                m_dSortedParticleIndex,
                                m_particleArray,
                                m_numParticles,
                                m_numGridCells);

    gpuErrchk( hipDeviceSynchronize());
    std::cout<<"------ REORDERING AND SORTING ------\n";
    for (int i = 0; i < 100; i++){
        float3 p = m_sortedParticleArray[i].position;
        int3 gridPos = calcGridPos(p, m_worldOrigin, m_cellSize);
        uint gridHash = calcGridHash(gridPos, m_gridSize);
        std::cout << "Particle "<<p.x<<", "<<p.y<<", "<<p.z
                    <<", GridPos: "<<gridPos.x<<", "<<gridPos.y<<", "<<gridPos.z<<", Hash: "<<gridHash
                    <<", Cell Start: "<<m_cellStart[gridHash]<<", Cell End: "<<m_cellEnd[gridHash]
                    <<", Index: "<<m_dSortedParticleIndex[i]<<std::endl;
    }

    //TODO: call calcSph

}

void ParticleSystem::checkNeighbors(uint index){

    Particle testParticle = m_particleArray[index];
    float3 p = testParticle.position;
    std::cout<<" ### NEIGHBORS FOR PARTICLE "<<p.x<<","<<p.y<<","<<p.z<<" ###\n"<<std::endl;
    
    std::vector<uint> neighbors;

    getNeighbors(p, m_dSortedParticleHash, m_dSortedParticleIndex, neighbors, numElements);
    printf("Without sorting: \n");
    for(int i = 0; i < neighbors.size(); i++){
        uint index = neighbors[i];
        float3 neighPos = m_particleArray[index].position;
        printf("Neighbor %d, pos (%f,%f,%f)\n", index, neighPos.x, neighPos.y, neighPos.z);
    }

    neighbors.clear();
    getSortedNeighbors(p, m_cellStart, m_cellEnd, neighbors, numElements);
    printf("\nWith sorting: \n");
    for(int i = 0; i < neighbors.size(); i++){
        uint index = neighbors[i];
        float3 neighPos = m_sortedParticleArray[index].position;
        printf("Neighbor %d, pos (%f,%f,%f)\n", index, neighPos.x, neighPos.y, neighPos.z);
    }
}

void ParticleSystem::_initParticles(int numParticles){
    std::random_device seed;
    std::default_random_engine rng(seed());
    std::uniform_real_distribution<float> pos(0, m_gridSize.x*m_cellSize.x); //assumes all dims to be same size
    
    for (auto it = m_particleArray; it != m_particleArray + numParticles; it++) {
        it->position = make_float3(pos(rng),pos(rng),pos(rng));
        it->velocity = make_float3(0.f,0.f,0.f);
        it->mass = 1.f;
    }
}

void ParticleSystem::_init(int numParticles){
   

    gpuErrchk(hipMallocManaged(&m_particleArray, numParticles * sizeof(Particle)));
    gpuErrchk(hipMallocManaged(&m_sortedParticleArray, numParticles * sizeof(Particle)));
    _initParticles(numParticles);

    // gpuErrchk(hipMallocManaged(&m_particles, numParticles * sizeof(float3)));
    // gpuErrchk(hipMallocManaged(&m_sortedParticles, numParticles * sizeof(float3)));
    // genRandomData(m_particles, m_particles + numParticles, m_gridSize.x*m_cellSize.x); //assumes all dims to be same size
    
    gpuErrchk(hipMallocManaged(&m_dGridParticleHash, numParticles * sizeof(uint)));
    gpuErrchk(hipMallocManaged(&m_dGridParticleIndex, numParticles * sizeof(uint)));

    gpuErrchk(hipMallocManaged(&m_dSortedParticleHash, numParticles * sizeof(uint)));
    gpuErrchk(hipMallocManaged(&m_dSortedParticleIndex, numParticles * sizeof(uint)));

    gpuErrchk(hipMallocManaged(&m_cellStart, m_numGridCells * sizeof(uint)));
    gpuErrchk(hipMallocManaged(&m_cellEnd, m_numGridCells * sizeof(uint)));
}

void ParticleSystem::_free(){
    hipFree(m_particleArray);
    hipFree(m_sortedParticleArray);
    hipFree(m_dGridParticleHash);
    hipFree(m_dGridParticleIndex);
    hipFree(m_dSortedParticleHash);
    hipFree(m_dSortedParticleIndex);
    hipFree(m_cellStart);
    hipFree(m_cellEnd);
}

int main() {

    float3 hostWorldOrigin = make_float3(0.f,0.f,0.f);
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(worldOrigin), &hostWorldOrigin, sizeof(float3)));
    float3 hostCellSize = make_float3(2.f,2.f,2.f);
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cellSize), &hostCellSize, sizeof(float3)));
    uint3  hostGridSize = make_uint3(64,64,64); // must be power of 2
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(gridSize), &hostGridSize, sizeof(uint3)));

    ParticleSystem* psystem = new ParticleSystem(numElements, hostWorldOrigin, hostGridSize, 1);

    psystem->update();

    //for testing purposes
    psystem->checkNeighbors(5);
    return 0;
}