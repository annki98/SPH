#include "hip/hip_runtime.h"
#include <iostream>
#include <filesystem>
#include <random>
#define _USE_MATH_DEFINES
#include <math.h>

#include "Engine/Defs.h"

class particle 
{
    public:
        float density;
        float pressure;
        glm::vec3 pressureGradient;
        float mass;
        float viscocity;

        glm::vec3 position;
        glm::vec3 velocity;
        glm::vec3 convecAccel;
};

float radius;
glm::vec3 gravity; //no bleeding idea how to represent gravity as a vector

//placeholders!
std::vector<particle> neighbours = {};
particle thisParticle;

void sphNaive()
{
    //equation (2): denstiy

    float resultDensity = 0.f;

    for (int j = 0; j < neighbours.size(); j++)
    {
        float temp = static_cast<float>(64 * M_PI * pow(radius, 9));

        glm::vec3 tempVector = thisParticle.position - neighbours[j].position;

        resultDensity += neighbours[j].mass * ( 315 / temp ) * glm::pow( (glm::pow(radius, 2) - glm::pow(glm::length(tempVector), 2)), 3);
    }
    thisParticle.density = resultDensity;

    //pressure

    float constantK = 1.f; //need to find out what the value of this goddamn constant is
    float restingDensity = 1.f;  //same for this thing
    float pressure = constantK * (thisParticle.density * restingDensity);

    //equation(3): pressure gradient

    glm::vec3 resultPG;

    for (int j = 0; j < neighbours.size(); j++)
    {
        float thisTemp = (thisParticle.pressure / pow(thisParticle.density, 2));

        float neighbourTemp = (neighbours[j].pressure / pow(neighbours[j].density, 2));

        glm::vec3 tempVector = thisParticle.position - neighbours[j].position;

        resultPG += (tempVector / glm::length(tempVector)) * static_cast<float>(neighbours[j].mass * ( thisTemp + neighbourTemp ) * ( -45 / (M_PI * pow(radius, 6))) * pow(radius - glm::length(tempVector), 2));
    }
    thisParticle.pressureGradient = resultPG;

    //equation(4): viscocity

    glm::vec3 resultV;

    for (int j = 0; j < neighbours.size(); j++)
    {
        glm:: vec3 tempVector = thisParticle.position - neighbours[j].position;

        resultV += ( ( neighbours[j].velocity - thisParticle.velocity) / neighbours[j].density)* static_cast<float>(neighbours[j].mass * ( 45 / (M_PI * pow(radius, 6))) * (radius - glm::length(tempVector)));   
    }
    glm::vec3 viscousTerm = (thisParticle.viscocity / thisParticle.density) * resultV;
    //equation (1) : acceleration

    glm::vec3 acceleration = gravity - thisParticle.pressureGradient + viscousTerm;
}